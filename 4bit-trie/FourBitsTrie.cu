#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

typedef struct entry
{
	unsigned int IP;
	unsigned char len;
	unsigned char nexthop;
}ENTRY;

typedef struct trie
{
	unsigned int* ArrayNode;
	unsigned int* ArrayNHP;
	unsigned char* ArrayLen;
}TRIE;

ENTRY* TableEntry;
unsigned int num_Entry = 0;
unsigned int num_Query = 0;
unsigned int NumNode = 0;

TRIE root;
unsigned int* D_ArrayNode;
unsigned int* D_ArrayNHP;
unsigned int* H_TableQuery;
unsigned int* H_TableQueryNHP;
unsigned int* D_TableQuery;
unsigned int* D_TableQueryNHP;

ENTRY ReadTable(char* str)
{
	char token[] = "./";
	char buf[256];
	char* strTemp;
	ENTRY entryTemp;
	
	sprintf(buf, "%s\0", strtok(str, token));
	entryTemp.IP = atoi(buf);
	sprintf(buf, "%s\0", strtok(NULL, token));
	entryTemp.IP = (entryTemp.IP << 8) + atoi(buf);
	sprintf(buf, "%s\0", strtok(NULL, token));
	entryTemp.IP = (entryTemp.IP << 8) + atoi(buf);
	sprintf(buf, "%s\0", strtok(NULL, token));
	entryTemp.IP = (entryTemp.IP << 8) + atoi(buf);
	entryTemp.nexthop = (entryTemp.IP >> 8) % 256;
	strTemp = (char* )strtok(NULL, token);
	
	unsigned int i;
	if (strTemp != NULL)
	{
		sprintf(buf, "%s\0", strTemp);
		entryTemp.len = atoi(buf);
	}
	else
	{
		if (entryTemp.IP >> 24 == 0 && (entryTemp.IP >> 16) % 256 == 0 && (entryTemp.IP >> 8) % 256 == 0 && entryTemp.IP % 256 == 0) entryTemp.len = 0;
		else if((entryTemp.IP >> 16) % 256 == 0 && (entryTemp.IP >> 8) % 256 == 0 && entryTemp.IP % 256 == 0)
		{
			i = 0;
			while(1)
			{
				if (entryTemp.IP >> 24 & (1 << i)) i++;
				else break;
			}
			entryTemp.len = 8 - i;
		}
		else if ((entryTemp.IP >> 8) % 256 == 0 && entryTemp.IP % 256 == 0)
		{
			i = 0;
			while(1)
			{
				if ((entryTemp.IP >> 8) % 256 & (1 << i)) i++;
				else break;
			}
			entryTemp.len = 16 - i;
		}
		else if (entryTemp.IP % 256 == 0)
		{
			i = 0;
			while(1)
			{
				if ((entryTemp.IP >> 8) % 256 & (1 << i)) i++;
				else break;
			}
			entryTemp.len = 24 - i;
		}		
		else
		{
			i = 0;
			while(1)
			{
				if (entryTemp.IP % 256 & (1 << i)) i++;
				else break;
			}
			entryTemp.len = 32 - i;
		}
	}
	return entryTemp;
}

void SetEntryTable(char* FileName)
{
	FILE* ipt = fopen(FileName, "r");
	char buf[256];
	
	num_Entry = 0;
	while(fgets(buf, 256, ipt) != NULL) num_Entry++;
	TableEntry = (ENTRY* )malloc(num_Entry * sizeof(ENTRY));
	
	rewind(ipt);
	num_Entry = 0;
	while(fgets(buf, 256, ipt) != NULL)
	{
		TableEntry[num_Entry] = ReadTable(buf);
		num_Entry++;
	}
	
	fclose(ipt);
	return;
}

TRIE CreateNode()
{
	unsigned int i;
	TRIE TrieTemp;
	if (NumNode == 0)
	{
		TrieTemp.ArrayNode = (unsigned int* )malloc(16 * sizeof(unsigned int));
		TrieTemp.ArrayNHP = (unsigned int* )malloc(sizeof(unsigned int));
		TrieTemp.ArrayLen = (unsigned char* )malloc(sizeof(unsigned char));
	}
	else
	{
		TrieTemp.ArrayNode = (unsigned int* )realloc(root.ArrayNode, 16 * (NumNode + 1) * sizeof(unsigned int));
		TrieTemp.ArrayNHP = (unsigned int* )realloc(root.ArrayNHP, (NumNode + 1) * sizeof(unsigned int));
		TrieTemp.ArrayLen = (unsigned char* )realloc(root.ArrayLen, (NumNode + 1) * sizeof(unsigned char));
	}
	for(i = 0; i < 16; i++) TrieTemp.ArrayNode[16 * NumNode + i] = 0;
	TrieTemp.ArrayNHP[NumNode] = 256;
	TrieTemp.ArrayLen[NumNode] = 0;
	NumNode++;
	return TrieTemp;
}

void AddNode(ENTRY info)
{
	unsigned int curPos = 0;
	unsigned int i, limit, remainder, tmp;
	
	remainder = info.len % 4;
	limit = info.len / 4;
	
	for(i = 0; i < limit; i++)
	{
		tmp = (info.IP >> (28 - i * 4)) % 16;
		if(root.ArrayNode[16 * curPos + tmp] == 0) 
		{
			root = CreateNode();
			root.ArrayNode[16 * curPos + tmp] = NumNode - 1;
		}
		curPos = root.ArrayNode[16 * curPos + tmp];
	}
	if (remainder)
	{
		tmp = (info.IP >> (28 - limit * 4)) % 16;
		unsigned int count = 1 << (4 - remainder);
		for(i = 0; i < count; i++)
		{
			if (root.ArrayNode[16 * curPos + tmp + i] == 0)
			{
				root = CreateNode();
				root.ArrayNode[16 * curPos + tmp + i] = NumNode - 1;
			}
			if (root.ArrayLen[root.ArrayNode[16 * curPos + tmp + i]] < info.len)
			{
				root.ArrayNHP[root.ArrayNode[16 * curPos + tmp + i]] = info.nexthop;
				root.ArrayLen[root.ArrayNode[16 * curPos + tmp + i]] = info.len;
			}
		}
	}
	else
	{
		root.ArrayNHP[curPos] = info.nexthop;
		root.ArrayLen[curPos] = info.len;
	}
	return;
}

void CreateTrie()
{
	unsigned int i = 0;
	root = CreateNode();
	//BeginTime = rdtsc();
	for(i = 0; i < num_Entry; i++)
	{
		if (TableEntry[i].len != 0) AddNode(TableEntry[i]);
		else 
		{
			if (NumNode == 0) root = CreateNode();
			root.ArrayNHP[0] = 0;
			root.ArrayLen[0] = 0;
		}
	}
	//EndTime = rdtsc();
	return;
}

void SetTrieToDevice()
{
	hipError_t error;
	error = hipMalloc((void** ) &D_ArrayNode, 16 * NumNode * sizeof(unsigned int));
	if (error != hipSuccess)
    {
        printf("hipMalloc D_ArrayNode returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	error = hipMemcpy(D_ArrayNode, root.ArrayNode, 16 * NumNode * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("hipMemcpy (D_ArrayNode, root.ArrayNode) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	error = hipMalloc((void** ) &D_ArrayNHP, NumNode * sizeof(unsigned int));
	if (error != hipSuccess)
    {
        printf("hipMalloc D_ArrayNHP returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	error = hipMemcpy(D_ArrayNHP, root.ArrayNHP, NumNode * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("hipMemcpy (D_ArrayNHP, root.ArrayNHP) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	return;
}

void SetQueryTable(char* FileName)
{
	FILE* ipt = fopen(FileName, "r");
	char buf[256];
	char str[256];
	
	num_Query = 0;
	while(fgets(buf, 256, ipt) != NULL) num_Query++;
	H_TableQuery = (unsigned int* )malloc(num_Query * sizeof(unsigned int));
	H_TableQueryNHP = (unsigned int* )malloc(num_Query * sizeof(unsigned int));
	
	rewind(ipt);
	num_Query = 0;
	while(fgets(buf, 256, ipt) != NULL)
	{
		sprintf(str, "%s\0", strtok(buf, "./"));
		H_TableQuery[num_Query] = atoi(str);
		sprintf(str, "%s\0", strtok(NULL, "./"));
		H_TableQuery[num_Query] = (H_TableQuery[num_Query] << 8) + atoi(str);
		sprintf(str, "%s\0", strtok(NULL, "./"));
		H_TableQuery[num_Query] = (H_TableQuery[num_Query] << 8) + atoi(str);
		sprintf(str, "%s\0", strtok(NULL, "./"));
		H_TableQuery[num_Query] = (H_TableQuery[num_Query] << 8) + atoi(str);
		H_TableQueryNHP[num_Query] = 256;
		num_Query++;
	}
	
	fclose(ipt);
	return;
}

void Shuffle()
{
	//ENTRY tmp;
	unsigned int tmp;
	unsigned int i, j;
	
	srand(time(0));
	
	for(i = 0; i < num_Query - 1; i++)
	{
		j = i + (rand() % (num_Query - i));
		tmp = H_TableQuery[i];
		H_TableQuery[i] = H_TableQuery[j];
		H_TableQuery[j] = tmp;
	}
	
	return;
}

void SetQueryTableToDevice()
{
	hipError_t error;
	error = hipMalloc((void** ) &D_TableQuery, num_Query * sizeof(unsigned int));
	if (error != hipSuccess)
    {
        printf("hipMalloc D_TableQuery returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	error = hipMemcpy((void** ) D_TableQuery, H_TableQuery, num_Query * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("hipMemcpy (D_TableQuery, H_TableQuery) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	error = hipMalloc((void** ) &D_TableQueryNHP, num_Query * sizeof(unsigned int));
	if (error != hipSuccess)
    {
        printf("hipMalloc D_TableQueryNHP returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	error = hipMemcpy((void** ) D_TableQueryNHP, H_TableQueryNHP, num_Query * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("hipMemcpy (D_TableQueryNHP, H_TableQueryNHP) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	return;
}

__global__ void Search(unsigned int* ArrayNode, unsigned int* ArrayNHP, unsigned int* TableQuery, unsigned int* ResultQuery, unsigned int N)
{
	const unsigned int tid_in_grid = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (tid_in_grid < N)
	{
		ResultQuery[tid_in_grid] = ArrayNHP[0];
		for(unsigned int i = 0, CurPos = 0, tmp; i < 8; i++)
		{
			tmp = (TableQuery[tid_in_grid] >> (28 - i * 4)) % 16;
			if (ArrayNode[CurPos * 16 + tmp] == 0) break;
			CurPos = ArrayNode[CurPos * 16 + tmp];
			if (ArrayNHP[CurPos] != 256) ResultQuery[tid_in_grid] = ArrayNHP[CurPos];
		}
	}
	__syncthreads();
}

int main(int argc, char** argv)
{
	if (argc != 3)
	{
		printf("Please execute the file as the following way:\n");
		printf("%s  routing_table_file_name  query_table_file_name\n", argv[0]);
		exit(1);
	}
	
	SetEntryTable(argv[1]);
	CreateTrie();
	SetTrieToDevice();

	SetQueryTable(argv[2]);
	Shuffle();
	SetQueryTableToDevice();
	
	dim3 grid((num_Query + 512 - 1) / 512, 1, 1);
	dim3 thread(512, 1, 1);
	
	// Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    hipError_t error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	Search<<< grid, thread>>>(D_ArrayNode, D_ArrayNHP, D_TableQuery, D_TableQueryNHP, num_Query);
	
	// Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	// Compute and print the performance
    float msecPerMatrixMul = msecTotal / num_Query;
	printf("Time= %.10f msec\n", msecPerMatrixMul);
	
	error = hipMemcpy(H_TableQueryNHP, D_TableQueryNHP, num_Query * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("hipMemcpy (H_TableQueryNHP, D_TableQueryNHP) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	
	free(H_TableQuery);
	free(H_TableQueryNHP);
	hipFree(D_ArrayNode);
	hipFree(D_ArrayNHP);
	hipFree(D_TableQuery);
	hipFree(D_TableQueryNHP);
	return 0;
}